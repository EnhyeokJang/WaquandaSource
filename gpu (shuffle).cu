#include "hip/hip_runtime.h"
// (c) Copyright Enhyeok Jang, Yonsei University, Seoul, Korea.
#include <iostream>
#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include "time.h"
#include "sys/time.h"
#include "unistd.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "omp.h"

__global__ void g2(float* A, int n) {
  int i = threadIdx.x + 1024*blockIdx.x;
  float x;
  int j=1;
  if(n<=32) {
    do {
      int i2 = (i&j)/j;
      A[i] = (__shfl_xor_sync(-1,A[i],j)+(1-2.0*i2)*(A[i]))/2.0;
      j *= 2;
    } while(j<n);
  }
  else if(n<=1024) {
    do {
      int i2 = (i&j)/j;
      A[i] = (__shfl_xor_sync(-1,A[i],j)+(1-2.0*i2)*(A[i]))/2.0;
      j *= 2;
    } while(j<32);
    do {
      x = A[i];
      __syncthreads();
      if((i&j)==0) {
        A[i] = (x + A[i+j])/2.0;
        A[i+j] = (x - A[i+j])/2.0;
      }
      __syncthreads();
      j *= 2;
    } while(j<=n);
  }
  else {
    do {
      x = A[i];
      __syncthreads();
      if((i&j)==0) {
        A[i] = (x + A[i+j])/2.0;
        A[i+j] = (x - A[i+j])/2.0;
      }
      __syncthreads();
      j *= 2;
    } while(j<=n);
  }
}

int main()
{
  int paq = 25;// address qubits
  const int a = paq;//# of address qubits
  int ia = 1;//# of LUT
  int pa = pow(2,paq);//# of address states
  int n = ia*pa;
  hipEvent_t startc, stopc, startg, stopg, startgm, stopgm;
  hipEventCreate(&startc);
  hipEventCreate(&stopc);
  hipEventCreate(&startg);
  hipEventCreate(&stopg);
  hipEventCreate(&startgm);
  hipEventCreate(&stopgm);
  float ms0 = 0.0;
  float ms1 = 0.0;
  float ms2 = 0.0;
  int cnt = 0;
  float pi = 3.14159265359;
  float* A;
  //float* B, *B2;
  int gr = 1;
  int bl = n;
  if(n>1024) {
    gr = n/1024;
    bl = 1024;
  }
  A = (float*)malloc(n * sizeof(float));
  int p[a];
  for(int i=0; i<a; i++) {
    p[i] = 0;
  }
  for(int i=0; i<n; i++) {
    A[i] = rand()%255;
  }
  
  //main calculation
  double size = n * sizeof(float);
  float* d_A;
  hipMalloc((void **) &d_A, size);
  dim3 dimGrid(gr, 1, 1);
  dim3 dimBlock(bl, 1, 1);
  hipEventRecord(startgm);
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipEventRecord(startg);
  for(int j=1; j<n; j*=2) {
    g2<<<dimGrid, dimBlock>>>(d_B, n);
  }
  hipEventRecord(stopg);
  hipEventSynchronize(stopg);
  hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
  hipEventRecord(stopgm);
  hipEventSynchronize(stopgm);
  hipFree(d_A);
  free(A);
  hipEventElapsedTime(&ms0, startc, stopc);
  hipEventDestroy(startc);
  hipEventDestroy(stopc);
  hipEventElapsedTime(&ms1, startg, stopg);
  hipEventDestroy(startg);
  hipEventDestroy(stopg);
  hipEventElapsedTime(&ms2, startgm, stopgm);
  hipEventDestroy(startgm);
  hipEventDestroy(stopgm);
  printf("\n\n----------------------------------\n");
	printf("CPU Computation time %lf ms\n\n", ms0);
	printf("GPU Computation time %lf ms\n\n", ms1);
	printf("GPU Computation + Host-Device Transfer time %lf ms\n\n", ms2);
}