#include <iostream>
#include "stdio.h"
#include <stdint.h>
#include "math.h"
#include "stdlib.h"
#include "time.h"
#include "sys/time.h"
#include "unistd.h"
#include "hip/hip_runtime.h"



__global__ void g2(float* A, long long n) {
  long long i = threadIdx.x;
  long long ji = 1024*threadIdx.x + blockIdx.x;
  float x = A[ji];
  long long j=1;
    do {
      x = A[i];
      __syncthreads();
      if((i&j)==0) {
        A[i] = (x + A[i+j])/2.0;
        A[i+j] = (x - A[i+j])/2.0;
      }
      __syncthreads();
      j *= 2;
    } while(j<n);
  //}
}

int main() {
  hipEvent_t start, stop, startg, stopg;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventCreate(&startg);
  hipEventCreate(&stopg);
  float ms0 = 0.0, ms1 = 0.0;
  long long n = 4294967296; // address states
  float* A;
  A = (float*)malloc(n * sizeof(float));
  double size = n * sizeof(float);
  float* d_A;
  for(long long i=0; i<n; i++) {
    A[i] = rand()%255;
  }
  hipMalloc((void **) &d_A, size);
  hipEventRecord(startg);
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipEventRecord(start);
  g2<<<n/1024, 1024>>>(d_A, n);
  hipEventRecord(stop);
  hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
  hipEventRecord(stopg);
  hipFree(d_A);
  free(A);
  hipEventElapsedTime(&ms0, start, stop);
  hipEventElapsedTime(&ms1, startg, stopg);
	printf("\n\nGPU warp_shuffle Computation time %lf ms\n\n", ms0);
  printf("\n\nGPU warp_shuffle Computation time with memcpy %lf ms\n\n", ms1);
}